#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>

unsigned int getmax(unsigned int *, unsigned int);


//my function to get the max number in the array using Nvdia parallel reduction techniques.
__global__ void getmaxcu(unsigned int* numbersDevice, unsigned int size, unsigned int* max){

    int threadID = threadIdx.x;
    int uniqueID = threadID + (blockDim.x * blockIdx.x);
//    printf("Checking tID:%d, bDimension:%d, blockID:%d, and id:%d.\n", threadIdx.x, blockDim.x, blockIdx.x, id);

    __syncthreads();
    for(int stride = 1; stride < size; stride *= 2){
	if(uniqueID % (stride * 2) == 0){
	    if(numbersDevice[uniqueID] < numbersDevice[uniqueID + stride]){
	    	numbersDevice[uniqueID] = numbersDevice[uniqueID + stride];
		if(numbersDevice[uniqueID + stride] > *max)
		    *max = numbersDevice[uniqueID + stride];
	    }
	    else{
		numbersDevice[uniqueID + stride] = numbersDevice[uniqueID];
	    }
	    if(numbersDevice[uniqueID] > *max)
		*max = numbersDevice[uniqueID];
	}
	__syncthreads();
    }

    if(uniqueID == 0){
	if(numbersDevice[0] > *max)
	    *max = numbersDevice[0];
    }
}

int main(int argc, char *argv[])
{
    unsigned int size = 0;  // The size of the array
    unsigned int i;  // loop index
    unsigned int * numbers; //pointer to the array
    unsigned int result[1];
    result[0] = 0;
 
    int numOfThreads;    
    int numOfBlocks;

    if(argc !=2)
    {
       printf("usage: maxseq num\n");
       printf("num = size of the array\n");
       exit(1);
    }
   
    size = atol(argv[1]);

    numbers = (unsigned int *)malloc(size * sizeof(unsigned int));
    if( !numbers )
    {
       printf("Unable to allocate mem for an array of size %u\n", size);
       exit(1);
    }    

    srand(time(NULL)); // setting a seed for the random number generator

    // Fill-up the array with random numbers from 0 to size-1 
    for( i = 0; i < size; i++){
       numbers[i] = rand()  % size;
//       printf("The number at numbers[%d]=%d.\n",i, numbers[i]);
    }

    //INFO ABOUT THE CURRENT DEVICE I AM USING
    hipDeviceProp_t dev;
    hipGetDeviceProperties(&dev, 0);
    numOfThreads = dev.maxThreadsPerBlock;
    //printf("size:%d and maxThreads:%d.\n", size, numOfThreads);

    if(size <= numOfThreads){
	numOfThreads = size;
	numOfBlocks = 1;
    }
    else{
	numOfBlocks = (int)ceil((double)size / (double)numOfThreads);
    }

    printf("Num of threads:%d and num of blocks:%d.\n", numOfThreads, numOfBlocks);
    unsigned int* numbersDevice;
    unsigned int* max;
    hipError_t error;

    error = hipMalloc((void**)&numbersDevice, size * sizeof(unsigned int));
    if(error != hipSuccess){
	printf("Error in cudaMalloc!!!!\n");
	exit(1);
    }
    error = hipMemcpy(numbersDevice, numbers, size * sizeof(unsigned int), hipMemcpyHostToDevice);
    if(error != hipSuccess){
	printf("Error in cudaMemcpy!!!!!\n");
	exit(1);
    }
    error = hipMalloc((void**)&max, sizeof(unsigned int));
    if(error != hipSuccess){
	printf("Error in hipMalloc for max.\n");
	exit(1);
    }
    error = hipMemcpy(max, result, sizeof(unsigned int), hipMemcpyHostToDevice);
    if(error != hipSuccess){
	printf("Error in hipMemcpy for max.\n");
	exit(1);
    }

    getmaxcu<<<numOfBlocks, numOfThreads>>>(numbersDevice, size, max);
    hipMemcpy(result, max, sizeof(unsigned int), hipMemcpyDeviceToHost);
    printf("Successfully finished the getmaxcu method and got max = %d.\n", *result);


    /*int nDevices;
    cudaGetDeviceCount(&nDevices);
    for(int k = 0; k < nDevices; k++){
	cudaDeviceProp prop;
        cudaGetDeviceProperties(&prop, k);
	printf("Device Number: %d\n", k);
	printf("Device Name: %s\n", prop.name);
	printf("Max threads per block: %d\n", prop.maxThreadsPerBlock);
	printf("Warp size: %d\n", prop.warpSize);
	printf("Num of Mps: %d\n", prop.multiProcessorCount);
	for(int z = 0; z < 3; z++){
	    printf("MaxthreadsDim: %d maxGridSize: %d\n", prop.maxThreadsDim[z], prop.maxGridSize[z]);
	    
	}
    }*/
   
//    printf("The maximum number in the array is: %u\n", 
//           getmax(numbers, size));

    hipFree(numbersDevice);
    hipFree(max);
    free(numbers);
    exit(0);
}


/*
   input: pointer to an array of long int
          number of elements in the array
   output: the maximum number of the array
*/
unsigned int getmax(unsigned int num[], unsigned int size)
{
  unsigned int i;
  unsigned int max = num[0];

  for(i = 1; i < size; i++)
	if(num[i] > max)
	   max = num[i];

  return( max );

}
